
#include <hip/hip_runtime.h>
/*******************************************************************************
*   OpenHyperFLOW2D-CUDA                                                       *
*                                                                              *
*   Transient, Density based Effective Explicit Parallel Hybrid Solver         *
*   TDEEPHS (CUDA+MPI)                                                         *
*   Version  1.0.1                                                             *
*   Copyright (C)  1995-2014 by Serge A. Suchkov                               *
*   Copyright policy: LGPL V3                                                  *
*                                                                              *
*   deeps2d_core.cpp: CUDA kernels code.                                       *
*                                                                              *
*  last update: 16/01/2014                                                     *
********************************************************************************/

#ifdef _CUDA_
#define _PARALLEL_ONLY

#include <cuda.h>
#include <cuda_runtime.h>
#include <cuda_runtime_api.h>

#include "libDEEPS2D/deeps2d_core.hpp"

__device__ int floatToOrderedInt( float floatVal ) {

int intVal = __float_as_int( floatVal );

return (intVal >= 0 ) ? intVal : intVal ^ 0x7FFFFFFF;

}

__device__  float orderedIntToFloat( int intVal ) {

return __int_as_float( (intVal >= 0) ? intVal : intVal ^ 0x7FFFFFFF );

}

__device__ float fatomicMin(float *addr, float value) {

        float old = *addr, assumed;

        if(old <= value) return old;

        do {
                assumed = old;
                old = atomicCAS((unsigned int*)addr, __float_as_int(assumed), __float_as_int(value));

        }while(old!=assumed);

        return old;
}

__device__ float fatomicMax(float *addr, float value) {

        float old = *addr, assumed;

        if(old >= value) return old;

        do {
                assumed = old;
                old = atomicCAS((unsigned int*)addr, __float_as_int(assumed), __float_as_int(value));

        }while(old!=assumed);

        return old;
}

void CUDA_BRRIER(char* KernelName) {
    cudaError_t cudaState = cudaDeviceSynchronize();    
    if(cudaState != cudaSuccess) {
        printf("\nError in %s kernel...\n",KernelName);
        if(cudaState == cudaErrorMemoryAllocation) {
           printf("Memory allocation error.\n");
        } else if(cudaState == cudaErrorLaunchTimeout ) {
          printf("Timeout.\n");
        }else if(cudaState == cudaErrorLaunchOutOfResources) {
          printf("Resources temporary insufficient.\n");
        }else if(cudaState ==  cudaErrorInvalidConfiguration ) {
          printf("Resources insufficient for this device\n");
        }else if(cudaState == cudaErrorInvalidValue) {
          printf("Invalid value.\n");
        }else if(cudaState == cudaErrorInvalidHostPointer ) {
          printf("Invalid host pointer.\n");
        }else if(cudaState == cudaErrorInvalidDevicePointer) {
          printf("Invalid device pointer.\n");
        }else if(cudaState == cudaErrorNotReady) {
          printf("Device not ready.\n");
        }else if(cudaState == cudaErrorLaunchFailure) {
          printf("Device illegal address.\n");
        } else {
         printf("Unknown error.\n");
        }
        Exit_OpenHyperFLOW2D();
    }
}

void CopyDeviceToDevice(void* src, void* dst, size_t length) {
    //printf("CopyDeviceToDevice: src=%x dst=%x length=%x\n",src,dst,length);
    cudaError_t cudaState = cudaMemcpyAsync(dst, src, length,cudaMemcpyDeviceToDevice);
    if(cudaState != cudaSuccess) {
     printf("\nError copy device to device...\n");
     if(cudaState == cudaErrorMemoryAllocation) {
        printf("Memory allocation error.\n");
     } else if(cudaState == cudaErrorLaunchTimeout ) {
       printf("Timeout.\n");
     }else if(cudaState == cudaErrorLaunchOutOfResources) {
       printf("Resources temporary insufficient.\n");
     }else if(cudaState ==  cudaErrorInvalidConfiguration ) {
       printf("Resources insufficient for this device\n");
     }else if(cudaState == cudaErrorInvalidValue) {
       printf("Invalid value.\n");
     }else if(cudaState == cudaErrorInvalidHostPointer ) {
       printf("Invalid host pointer.\n");
     }else if(cudaState == cudaErrorInvalidDevicePointer) {
       printf("Invalid device pointer.\n");
     }else if(cudaState == cudaErrorNotReady) {
       printf("Device not ready.\n");
     } else {
      printf("Unknown error.\n");
     }
       Exit_OpenHyperFLOW2D();
    }
}

void CopyHostToDevice(void* src, void* dst, size_t length) {
    cudaError_t cudaState = cudaMemcpyAsync(dst, src, length,cudaMemcpyHostToDevice);
    if(cudaState != cudaSuccess) {
     printf("\nError copy host to device...\n");
     if(cudaState == cudaErrorMemoryAllocation) {
        printf("Memory allocation error.\n");
     } else if(cudaState == cudaErrorLaunchTimeout ) {
       printf("Timeout.\n");
     }else if(cudaState == cudaErrorLaunchOutOfResources) {
       printf("Resources temporary insufficient.\n");
     }else if(cudaState ==  cudaErrorInvalidConfiguration ) {
       printf("Resources insufficient for this device\n");
     }else if(cudaState == cudaErrorInvalidValue) {
       printf("Invalid value.\n");
     }else if(cudaState == cudaErrorInvalidHostPointer ) {
       printf("Invalid host pointer.\n");
     }else if(cudaState == cudaErrorInvalidDevicePointer) {
       printf("Invalid device pointer.\n");
     }else if(cudaState == cudaErrorNotReady) {
       printf("Device not ready.\n");
     } else {
      printf("Unknown error.\n");
     }
       Exit_OpenHyperFLOW2D();
    }
}
     
void CopyDeviceToHost(void* src, void* dst, size_t length) {
    cudaError_t cudaState = cudaMemcpyAsync(dst, src, length,cudaMemcpyDeviceToHost);
    if(cudaState != cudaSuccess) {
     printf("\nError copy device to host...\n");
     if(cudaState == cudaErrorMemoryAllocation) {
        printf("Memory allocation error.\n");
     } else if(cudaState == cudaErrorLaunchTimeout ) {
       printf("Timeout.\n");
     }else if(cudaState == cudaErrorLaunchOutOfResources) {
       printf("Resources temporary insufficient.\n");
     }else if(cudaState ==  cudaErrorInvalidConfiguration ) {
       printf("Resources insufficient for this device\n");
     }else if(cudaState == cudaErrorInvalidValue) {
       printf("Invalid value.\n");
     }else if(cudaState == cudaErrorInvalidHostPointer ) {
       printf("Invalid host pointer.\n");
     }else if(cudaState == cudaErrorInvalidDevicePointer) {
       printf("Invalid device pointer.\n");
     }else if(cudaState == cudaErrorNotReady) {
       printf("Device not ready.\n");
     } else {
      printf("Unknown error.\n");
     }
       Exit_OpenHyperFLOW2D();
    }
}

__global__ void 
cuda_SetInitBoundaryLayer(FlowNode2D<double,NUM_COMPONENTS>* pJ2D,
                          unsigned long int index_limit,
                          int X0, int MAX_Y,
                          double delta,
                          double sig_w, 
                          double sig_f,
                          TurbulenceExtendedModel etm,
                          double _dx, double _dy,
                          double* _Hu,
                          int _isSrcAdd,
                          FlowType _FT) {

    unsigned long int index = threadIdx.x + blockIdx.x * blockDim.x;

    if(index < index_limit) {

              FlowNode2D< double,NUM_COMPONENTS >* CurrentNode=&pJ2D[index];
              
              if(CurrentNode->CT != (ulong)(CT_SOLID_2D | CT_NODE_IS_SET_2D)) {   
                  
                  if(CurrentNode->time == 0. &&  delta > 0.0  &&  CurrentNode->l_min <= delta) {
                     CurrentNode->S[i2d_RoU] = CurrentNode->S[i2d_RoU] * CurrentNode->l_min/delta;
                     CurrentNode->S[i2d_RoV] = CurrentNode->S[i2d_RoV] * CurrentNode->l_min/delta;
                     CurrentNode->FillNode2D(0,1,sig_w,sig_f,etm,delta,_dx,_dy,_Hu,_isSrcAdd,_FT);
                  }
              
               if(CurrentNode->CT != (ulong)(NT_FC_2D)) { 

                  int  i = CurrentNode->ix - X0;
                  int  j = CurrentNode->iy;

                  int  n1 = CurrentNode->idXl; 
                  int  n2 = CurrentNode->idXr;
                  int  n3 = CurrentNode->idYu;
                  int  n4 = CurrentNode->idYd;

                  int  N1 = i - n1;
                  int  N2 = i + n2;
                  int  N3 = j + n3;
                  int  N4 = j - n4;
                  
                  CurrentNode->UpNode    = &pJ2D[i*MAX_Y + N3];
                  CurrentNode->DownNode  = &pJ2D[i*MAX_Y + N4];
                  CurrentNode->RightNode = &pJ2D[N2*MAX_Y + j];
                  CurrentNode->LeftNode  = &pJ2D[N1*MAX_Y + j];
               }
              }
   }
}
                                                                     
__global__ void
cuda_SetMinDistanceToWall2D(FlowNode2D<double,NUM_COMPONENTS>* pJ2D,
                            unsigned long int index_limit,
                            XY<int>* WallNodes2D, 
                            int NumWallNodes2D,
                            double min_l_min,
                            double max_l_min,
                            double _dx, double _dy)   {
          
   unsigned long int index = threadIdx.x + blockIdx.x * blockDim.x;
   
   if(index < index_limit) {
       
       
       FlowNode2D<double,NUM_COMPONENTS>* TmpNode = &pJ2D[index];
       
       if(TmpNode->CT != (ulong)(CT_SOLID_2D | CT_NODE_IS_SET_2D)) {   
          
           TmpNode->l_min = max_l_min;
 #pragma unroll
           for (int ii=0;ii<NumWallNodes2D;ii++) {

              XY<int>*  TmpWallNode = &WallNodes2D[ii]; 

              double L_x   = (TmpWallNode->X - TmpNode->ix)* _dx;
              double L_y   = (TmpWallNode->Y - TmpNode->iy)* _dy;
              double l_min = sqrt(L_x*L_x + L_y*L_y);
              
              TmpNode->l_min = max(min(TmpNode->l_min,l_min),min_l_min);
            }
        }
   }
   //__syncthreads();
}
__global__ void 
cuda_Recalc_y_plus(FlowNode2D<double,NUM_COMPONENTS>* pJ2D,
                   unsigned long int index_limit,
                   XY<int>* WallNodes2D, 
                   int NumWallNodes2D,
                   double min_l_min,
                   double max_l_min,
                   double _dx, 
                   double _dy,
                   int max_y) {
    unsigned long int index = threadIdx.x + blockIdx.x * blockDim.x;

    if(index < index_limit) {

        FlowNode2D<double,NUM_COMPONENTS>* TmpNode = &pJ2D[index];

        if(TmpNode->CT != (ulong)(CT_SOLID_2D | CT_NODE_IS_SET_2D)) {

            int iw=0;
            int jw=0;
#pragma unroll
           for (int ii=0;ii<NumWallNodes2D;ii++) {
               
               XY<int>*  TmpWallNode = &WallNodes2D[ii]; 

               double L_x   = (TmpWallNode->X - TmpNode->ix)* _dx;
               double L_y   = (TmpWallNode->Y - TmpNode->iy)* _dy;
               double l_min = sqrt(L_x*L_x + L_y*L_y);

               if(l_min  == TmpNode->l_min) {
                  iw = TmpWallNode->X;
                  jw = TmpWallNode->Y;
               }
           }
           
          //__syncthreads();

           unsigned long int wall_index = iw*max_y + jw;

           if(wall_index < index_limit) {
               
               FlowNode2D<double,NUM_COMPONENTS>* WallNode = &pJ2D[wall_index];   // x*nY + y

               double tau_w = (fabs(WallNode->dUdy) +                           
                               fabs(WallNode->dVdx)) * WallNode->mu;  

               double U_w   = sqrt(tau_w/WallNode->S[i2d_Ro]);

               TmpNode->y_plus = U_w*TmpNode->l_min*TmpNode->S[i2d_Ro]/TmpNode->mu;
           }
        }
     }  
  //__syncthreads(); 
}

__global__  void  
cuda_DEEPS2D_Stage1(FlowNode2D<double,NUM_COMPONENTS>*     pLJ,
                    FlowNodeCore2D<double,NUM_COMPONENTS>* pLC,
                    int MAX_X, int MAX_Y,// int X0, 
                    unsigned long r_limit,
                    unsigned long l_limit,
                    double dxx, double dyy,
                    double dtdx, double dtdy,
                    double _dt,
                    int _FT, int Num_Eq) {
    
    unsigned long int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    if((index >= l_limit) &&
       (index < r_limit)) {

          FlowNode2D< double,NUM_COMPONENTS >* CurrentNode=&pLJ[index];

          if(CurrentNode->CT != (ulong)(CT_SOLID_2D | CT_NODE_IS_SET_2D) &&
             CurrentNode->CT != (ulong)(NT_FC_2D)) { 

              FlowNodeCore2D< double,NUM_COMPONENTS >* NextNode=&pLC[index];
              
              double beta[NUM_COMPONENTS+6];  
              double _beta[NUM_COMPONENTS+6]; 
              
              int  n1 = CurrentNode->idXl; 
              int  n2 = CurrentNode->idXr;
              int  n3 = CurrentNode->idYu;
              int  n4 = CurrentNode->idYd;
              
              double  n_n_1 = 1./max(n1+n2,1);
              double  m_m_1 = 1./max(n3+n4,1);
              
              FlowNode2D< double,NUM_COMPONENTS >* UpNode    = CurrentNode->UpNode;
              FlowNode2D< double,NUM_COMPONENTS >* DownNode  = CurrentNode->DownNode;
              FlowNode2D< double,NUM_COMPONENTS >* RightNode = CurrentNode->RightNode;
              FlowNode2D< double,NUM_COMPONENTS >* LeftNode  = CurrentNode->LeftNode;
              
              // Scan equation system ... k - number of equation
#pragma unroll              
              for (int k=0;k<Num_Eq;k++ ) {
                    
                  int      c_flag = 0;
                  int      dx_flag, dx2_flag;                 

                  int      dy_flag, dy2_flag;
                  double   dXX,dYY;

                  beta[k]  = CurrentNode->beta[k];
                  _beta[k] = 1. - beta[k];

                // Precomputed variables for current node ...
                    c_flag  = dx_flag = dy_flag = dx2_flag = dy2_flag = 0;
                    if ( k < 4 ) { // Make bit flags for future test for current equation
                        c_flag   = CT_Ro_CONST_2D     << k; 
                        dx_flag  = CT_dRodx_NULL_2D   << k;
                        dy_flag  = CT_dRody_NULL_2D   << k;
                        dx2_flag = CT_d2Rodx2_NULL_2D << k;
                        dy2_flag = CT_d2Rody2_NULL_2D << k;
                    } else if (k < (4+NUM_COMPONENTS)) {
                        c_flag   = CT_Y_CONST_2D;
                        dx_flag  = CT_dYdx_NULL_2D;
                        dy_flag  = CT_dYdy_NULL_2D;
                        dx2_flag = CT_d2Ydx2_NULL_2D;
                        dy2_flag = CT_d2Ydy2_NULL_2D;
                    } else if ((CurrentNode->isTurbulenceCond2D(TCT_k_eps_Model_2D) ||
                                CurrentNode->isTurbulenceCond2D(TCT_Spalart_Allmaras_Model_2D))) { //
                      if( k == i2d_k) {
                          c_flag   = TCT_k_CONST_2D     << (k-4-NUM_COMPONENTS); 
                          dx_flag  = TCT_dkdx_NULL_2D   << (k-4-NUM_COMPONENTS);
                          dy_flag  = TCT_dkdy_NULL_2D   << (k-4-NUM_COMPONENTS);
                          dx2_flag = TCT_d2kdx2_NULL_2D << (k-4-NUM_COMPONENTS);
                          dy2_flag = TCT_d2kdy2_NULL_2D << (k-4-NUM_COMPONENTS);
                      } else if (k == i2d_eps) {
                          c_flag   = TCT_eps_CONST_2D     << (k-4-NUM_COMPONENTS); 
                          dx_flag  = TCT_depsdx_NULL_2D   << (k-4-NUM_COMPONENTS);
                          dy_flag  = TCT_depsdy_NULL_2D   << (k-4-NUM_COMPONENTS);
                          dx2_flag = TCT_d2epsdx2_NULL_2D << (k-4-NUM_COMPONENTS);
                          dy2_flag = TCT_d2epsdy2_NULL_2D << (k-4-NUM_COMPONENTS);
                      }
                    }
                    // Check BC for current equation
                    if (k<(4+NUM_COMPONENTS)) {

                        if ( CurrentNode->isCond2D((CondType2D)c_flag) )
                            c_flag  = 0;
                        else
                            c_flag  = 1;

                        if ( CurrentNode->isCond2D((CondType2D)dx_flag) ) {
                            dx_flag = 0;
                        } else {
                            dx_flag = 1;
                        }

                        if ( CurrentNode->isCond2D((CondType2D)dy_flag) ) {
                            dy_flag = 0;
                        } else {
                            dy_flag = 1;
                        }

                        if ( CurrentNode->isCond2D((CondType2D)dx2_flag) ) {
                            dx2_flag = 1;
                        } else {
                            dx2_flag = 0;
                        }

                        if ( CurrentNode->isCond2D((CondType2D)dy2_flag) ) {
                            dy2_flag = 1;
                        } else {
                            dy2_flag = 0;
                        }
                    } else if((CurrentNode->isTurbulenceCond2D(TCT_k_eps_Model_2D) ||
                               CurrentNode->isTurbulenceCond2D(TCT_Spalart_Allmaras_Model_2D)) ) {
                        if ( CurrentNode->isTurbulenceCond2D((TurbulenceCondType2D)c_flag) )
                            c_flag  = 0;
                        else
                            c_flag  = 1;

                        if ( CurrentNode->isTurbulenceCond2D((TurbulenceCondType2D)dx_flag) ) {
                            dx_flag = 0;
                        } else {
                            dx_flag = 1;
                        }
                        if ( CurrentNode->isTurbulenceCond2D((TurbulenceCondType2D)dy_flag) ) {
                            dy_flag = 0;
                        } else {
                            dy_flag = 1;
                        }
                        if ( CurrentNode->isTurbulenceCond2D((TurbulenceCondType2D)dx2_flag) ) {
                            dx2_flag = 1;
                        } else {
                            dx2_flag = 0;
                        }
                        if ( CurrentNode->isTurbulenceCond2D((TurbulenceCondType2D)dy2_flag) ) {
                            dy2_flag = 1;
                        } else {
                            dy2_flag = 0;
                        }
                    }
                    
                    if ( c_flag ) {
                        if ( dx_flag ) {
                            dXX = (RightNode->A[k]-LeftNode->A[k])*n_n_1;  // CurrentNode->dSdx[k] = 
                        } else {
                            CurrentNode->S[k] = (LeftNode->S[k]*n2+RightNode->S[k]*n1)*n_n_1;
                            dXX = 0.;                                     //  CurrentNode->dSdx[k]=  
                        }
                        if ( dy_flag ) {
                            dYY = (UpNode->B[k]-DownNode->B[k])*m_m_1;    // CurrentNode->dSdy[k] =
                        } else {                                                                                                                    
                            CurrentNode->S[k] =  (UpNode->S[k]*n3+DownNode->S[k]*n4)*m_m_1;
                            dYY = 0;                                       // CurrentNode->dSdy[k] =
                        }
                        
                        // Cauchi BC temporary blocked 
                        /*
                        if ( dx2_flag ) {
                            dXX = (LeftNode->dSdx[k]+RightNode->dSdx[k])*0.5;
                        }
                        if ( dy2_flag ) {
                            dYY = (UpNode->dSdy[k]+DownNode->dSdy[k])*0.5;
                        }
                        */
                        if ( _FT ) {
                            NextNode->S[k] = CurrentNode->S[k]*beta[k]+_beta[k]*(dxx*(LeftNode->S[k]+RightNode->S[k])+dyy*(UpNode->S[k]+DownNode->S[k]))*0.5
                                          - (dtdx*dXX+dtdy*(dYY+CurrentNode->F[k]/(CurrentNode->ix+0.5))) + (CurrentNode->Src[k])*_dt+CurrentNode->SrcAdd[k];
                        } else {
                            NextNode->S[k] = CurrentNode->S[k]*beta[k]+_beta[k]*(dxx*(LeftNode->S[k]+RightNode->S[k])+dyy*(UpNode->S[k]+DownNode->S[k]))*0.5
                                          - (dtdx*dXX+dtdy*dYY) + (CurrentNode->Src[k])*_dt+CurrentNode->SrcAdd[k];
                        }
                }
            }
            //CalcChemicalReactions(CurrentNode,CRM_ZELDOVICH, (void*)(pCRMD));
          }
            
     }
}

__global__  void 
cuda_DEEPS2D_Stage2(FlowNode2D<double,NUM_COMPONENTS>*     pLJ,
                    FlowNodeCore2D<double,NUM_COMPONENTS>* pLC,
                    int ii, int MAX_X, int MAX_Y, //int X0, 
                    unsigned long r_limit,
                    unsigned long l_limit,
                    double beta_init, double  beta0, 
                    int b_FF, double CFL0,
                    ChemicalReactionsModelData2D* pCRMD,
                    int noTurbCond,
                    double SigW, double SigF, double dx_1, double dy_1, double delta_bl,
                    FlowType _FT, int Num_Eq,
#ifdef _RMS_             
                    double*  RMS, 
                    int*     iRMS,
                    double   DD_max,
                    int*     i_c,
                    int*     j_c,
#endif // _RMS_
                    double* _Hu,
                    int _isSrcAdd,
                    float* dt_global,
                    TurbulenceExtendedModel TurbExtModel ) {
    

    unsigned long int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    if((index >= l_limit) &&
       (index < r_limit)) {
        
          FlowNode2D< double,NUM_COMPONENTS >* CurrentNode=&pLJ[index];

          if(CurrentNode->CT != (ulong)(CT_SOLID_2D | CT_NODE_IS_SET_2D) &&
             CurrentNode->CT != (ulong)(NT_FC_2D)) { 
              
              double  beta_min;
              
              beta_min = min(beta0,beta_init);

              FlowNodeCore2D< double,NUM_COMPONENTS >* NextNode=&pLC[index];
              
              int  n1 = CurrentNode->idXl; 
              int  n2 = CurrentNode->idXr;
              int  n3 = CurrentNode->idYu;
              int  n4 = CurrentNode->idYd;
              
              double  n_n_1 = 1./max(n1+n2,1);
              double  m_m_1 = 1./max(n3+n4,1);
              
              FlowNode2D< double,NUM_COMPONENTS >* UpNode    = CurrentNode->UpNode;
              FlowNode2D< double,NUM_COMPONENTS >* DownNode  = CurrentNode->DownNode;
              FlowNode2D< double,NUM_COMPONENTS >* RightNode = CurrentNode->RightNode;
              FlowNode2D< double,NUM_COMPONENTS >* LeftNode  = CurrentNode->LeftNode;
              
              double dx_1xn_n_1=dx_1*n_n_1;
              double dy_1xm_m_1=dy_1*m_m_1;

              // Scan equation system ... k - number of equation
#pragma unroll              
              for (int k=0;k<Num_Eq;k++ ) {
                    
                  int      c_flag = 0;

                  if ( k < 4 ) // Make bit flags for future test for current equation 
                      c_flag  = CT_Ro_CONST_2D   << k;
                  else if (k<(4+NUM_COMPONENTS))  // 7 ?
                      c_flag  = CT_Y_CONST_2D;
                  else if((CurrentNode->isTurbulenceCond2D(TCT_k_eps_Model_2D) ||
                           CurrentNode->isTurbulenceCond2D(TCT_Spalart_Allmaras_Model_2D) )) 
                      c_flag  = TCT_k_CONST_2D << (k-4-NUM_COMPONENTS); 

                  if ( !CurrentNode->isCond2D((CondType2D)c_flag) && 
                        CurrentNode->S[k] != 0. ) {
                        double DD_local;
                        double Tmp;

                        if(k == i2d_RoU && k == i2d_RoV ) {
                            Tmp = sqrt(CurrentNode->S[i2d_RoU]*CurrentNode->S[i2d_RoU]+
                                       CurrentNode->S[i2d_RoV]*CurrentNode->S[i2d_RoV]+1.e-30); // Flux
                        } else {
                            Tmp = CurrentNode->S[k];
                        }

                        if(fabs(Tmp) > 1.e-15)
                           DD_local = fabs((NextNode->S[k]-CurrentNode->S[k])/Tmp);
                        else
                           DD_local = 0.0;
                        
                        
                        if( b_FF == BFF_L) {
                         //LINEAR locally adopted blending factor function  (LLABFF)
                           CurrentNode->beta[k] = min(beta_min,(beta_min*beta_min)/(beta_min+DD_local));
                         } else if( b_FF == BFF_LR) {
                         //LINEAR locally adopted blending factor function with relaxation (LLABFFR)
                           CurrentNode->beta[k] = min((beta_min+CurrentNode->beta[k])*0.5,(beta_min*beta_min)/(beta_min+DD_local));
                         } else if( b_FF == BFF_S) {
                         //SQUARE locally adopted blending factor function (SLABF)
                           CurrentNode->beta[k] = min(beta_min,(beta_min*beta_min)/(beta_min+DD_local*DD_local));
                         } else if (b_FF == BFF_SR) {
                         //SQUARE locally adopted blending factor function with relaxation (SLABFFR)
                           CurrentNode->beta[k] = min((beta_min+CurrentNode->beta[k])*0.5,(beta_min*beta_min)/(beta_min+DD_local*DD_local));
                         } else if( b_FF == BFF_SQR) {
                         //SQRT() locally adopted blending factor function (SQRLABF)
                           CurrentNode->beta[k] = min(beta_min,(beta_min*beta_min)/(beta_min+sqrt(DD_local)));
                         } else if( b_FF == BFF_SQRR) {
                         //SQRT() locally adopted blending factor function with relaxation (SQRLABFFR)
                           CurrentNode->beta[k] = min((beta_min+CurrentNode->beta[k])*0.5,(beta_min*beta_min)/(beta_min+sqrt(DD_local))); 
                         } else {
                           // Default->SQRLABF
                           CurrentNode->beta[k] = min(beta_min,(beta_min*beta_min)/(beta_min+sqrt(DD_local)));
                         }
                         
#ifdef _RMS_
                         RMS[k+ii*Num_Eq] += DD_local;
                         iRMS[k+ii*Num_Eq]++;
                         DD_max[k+ii*Num_Eq] = max(DD_max[k+ii*Num_Eq],DD_local);

                         if ( DD_max[k+ii*Num_Eq] == DD_local ) {
                              i_c[ii] = i;
                              j_c[ii] = j;
                         }
#endif // RMS
                  }
                  
                  if (k<(4+NUM_COMPONENTS)) {
                      if ( !CurrentNode->isCond2D((CondType2D)c_flag) )
                            CurrentNode->S[k]   = NextNode->S[k];
                  } else if ((CurrentNode->isTurbulenceCond2D(TCT_k_eps_Model_2D) ||
                              CurrentNode->isTurbulenceCond2D(TCT_Spalart_Allmaras_Model_2D)) ){
                      if ( !CurrentNode->isTurbulenceCond2D((TurbulenceCondType2D)c_flag) )
                            CurrentNode->S[k]   =  NextNode->S[k];
                  }
                  
              }

              CurrentNode->droYdx[NUM_COMPONENTS]=CurrentNode->droYdy[NUM_COMPONENTS]=0.;

#pragma unroll
              for (int k=4;k<NUM_EQ-2;k++ ) {
                  if ( !CurrentNode->isCond2D(CT_dYdx_NULL_2D) ) {
                      CurrentNode->droYdx[k-4]=(RightNode->S[k]-LeftNode->S[k])*dx_1*0.5;
                      CurrentNode->droYdx[NUM_COMPONENTS]+=(RightNode->S[k]-LeftNode->S[k])*dx_1*0.5;
                  }
                  if ( !CurrentNode->isCond2D(CT_dYdy_NULL_2D) ) {
                        CurrentNode->droYdy[k-4]=(UpNode->S[k]-DownNode->S[k])*dy_1*0.5;
                        CurrentNode->droYdy[NUM_COMPONENTS]+=(DownNode->S[k]-UpNode->S[k])*dy_1*0.5;
                  }
              }
              
              if (CurrentNode->isCond2D(CT_WALL_NO_SLIP_2D) || CurrentNode->isCond2D(CT_WALL_LAW_2D) )  {
                  CurrentNode->dUdx=(RightNode->U*n1-LeftNode->U*n2)*dx_1xn_n_1;
                  CurrentNode->dVdx=(RightNode->V*n1-LeftNode->V*n2)*dx_1xn_n_1;

                  CurrentNode->dUdy=(UpNode->U*n3-DownNode->U*n4)*dy_1xm_m_1;
                  CurrentNode->dVdy=(UpNode->V*n3-DownNode->V*n4)*dy_1xm_m_1;

                  if(CurrentNode->isTurbulenceCond2D(TCT_k_eps_Model_2D)){
                    CurrentNode->dkdx   =(RightNode->S[i2d_k]*n1-LeftNode->S[i2d_k]*n2)*dx_1/CurrentNode->S[i2d_Ro]*n_n_1;
                    CurrentNode->depsdx =(RightNode->S[i2d_eps]*n1-LeftNode->S[i2d_eps]*n2)*dx_1/CurrentNode->S[i2d_Ro]*n_n_1;

                    CurrentNode->dkdy   =(UpNode->S[i2d_k]*n3-DownNode->S[i2d_k]*n4)*dy_1/CurrentNode->S[i2d_Ro]*m_m_1;
                    CurrentNode->depsdy =(UpNode->S[i2d_eps]*n3-DownNode->S[i2d_eps]*n4)*dy_1/CurrentNode->S[i2d_Ro]*m_m_1;
                  } else if (CurrentNode->isTurbulenceCond2D(TCT_Spalart_Allmaras_Model_2D)) {
                             CurrentNode->dkdx   =(RightNode->S[i2d_k]*n1-LeftNode->S[i2d_k]*n2)*dx_1/CurrentNode->S[i2d_Ro]*n_n_1;
                             CurrentNode->dkdy   =(UpNode->S[i2d_k]*n3-DownNode->S[i2d_k]*n4)*dy_1/CurrentNode->S[i2d_Ro]*m_m_1;
                  }
              } else {
                  CurrentNode->dUdx   =(RightNode->U-LeftNode->U)*dx_1xn_n_1;
                  CurrentNode->dVdx   =(RightNode->V-LeftNode->V)*dx_1xn_n_1;

                  CurrentNode->dUdy   =(UpNode->U-DownNode->U)*dy_1xm_m_1;
                  CurrentNode->dVdy   =(UpNode->V-DownNode->V)*dy_1xm_m_1;
                  if(CurrentNode->isTurbulenceCond2D(TCT_k_eps_Model_2D)){
                    CurrentNode->dkdx   =(RightNode->S[i2d_k]-LeftNode->S[i2d_k])*dx_1/CurrentNode->S[i2d_Ro]*n_n_1;
                    CurrentNode->depsdx =(RightNode->S[i2d_eps]-LeftNode->S[i2d_eps])*dx_1/CurrentNode->S[i2d_Ro]*n_n_1;

                    CurrentNode->dkdy   =(UpNode->S[i2d_k]-DownNode->S[i2d_k])*dy_1/CurrentNode->S[i2d_Ro]*m_m_1;
                    CurrentNode->depsdy =(UpNode->S[i2d_eps]-DownNode->S[i2d_eps])*dy_1/CurrentNode->S[i2d_Ro]*m_m_1;
                  } else if (CurrentNode->isTurbulenceCond2D(TCT_Spalart_Allmaras_Model_2D)) {
                             CurrentNode->dkdx   =(RightNode->S[i2d_k]-LeftNode->S[i2d_k])*dx_1/CurrentNode->S[i2d_Ro]*n_n_1;
                             CurrentNode->dkdy   =(UpNode->S[i2d_k]-DownNode->S[i2d_k])*dy_1/CurrentNode->S[i2d_Ro]*m_m_1;
                  }
              }

              CurrentNode->dTdx=(RightNode->Tg-LeftNode->Tg)*dx_1xn_n_1;
              CurrentNode->dTdy=(UpNode->Tg-DownNode->Tg)*dy_1xm_m_1;
              
              //CalcChemicalReactions(CurrentNode,CRM_ZELDOVICH, (void*)(pCRMD));
              
              if(noTurbCond) {
                 CurrentNode->FillNode2D(0,1,SigW,SigF,TurbExtModel,delta_bl,1.0/dx_1,1.0/dy_1,_Hu,_isSrcAdd,_FT);
              } else {
                 CurrentNode->FillNode2D(1,0,SigW,SigF,TurbExtModel,delta_bl,1.0/dx_1,1.0/dy_1,_Hu,_isSrcAdd,_FT);
              }
              
              if( CurrentNode->Tg < 0. ) {
                  *dt_global = 0.0;  // Computational instability
              }  else {
                  double AAA          = sqrt(CurrentNode->k*CurrentNode->R*CurrentNode->Tg); 
                  double dt_min_local = CFL0*min(dx_1*(AAA+fabs(CurrentNode->U)),dy_1*(AAA+fabs(CurrentNode->V)));
                  //*dt_global = min(*dt_global,dt_min_local);
                  fatomicMin(dt_global,dt_min_local);
              }
       }
     }
   }
#endif // _CUDA_

